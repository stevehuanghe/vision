#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

template <typename T>
__device__ T bilinear_interpolate(
    const T* input,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }
  T eps = 0.;
  T ly = y - y_low + eps;
  T lx = x - x_low + eps;
  T hy = 1. - ly, hx = 1. - lx;

  // do bilinear interpolation
  T v1 = input[y_low * width + x_low];
  T v2 = input[y_low * width + x_high];
  T v3 = input[y_high * width + x_low];
  T v4 = input[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForward(
    const int nthreads,
    const T* input,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* rois,
    T* output) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_rois[1] * spatial_scale;
    T roi_start_h = offset_rois[2] * spatial_scale;
    T roi_end_w = offset_rois[3] * spatial_scale;
    T roi_end_h = offset_rois[4] * spatial_scale;

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_input =
        input + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(offset_input, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    output[index] = output_val;
  }
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const T* input,
    const int height,
    const int width,
    T y,
    T x,
    T& w1,
    T& w2,
    T& w3,
    T& w4,
    T& w_x,
    T& w_y,
    int& x_low,
    int& x_high,
    int& y_low,
    int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }
  T eps = 0.;
  T ly = y - y_low + eps;
  T lx = x - x_low + eps;
  T hy = 1. - ly, hx = 1. - lx;

  // f(x_q, y_q)
  T f1 = input[y_low * width + x_low];
  T f2 = input[y_low * width + x_high];
  T f3 = input[y_high * width + x_low];
  T f4 = input[y_high * width + x_high];

  T g1_x = -hx;
  T g2_x = -lx;
  T g3_x = hx;
  T g4_x = lx;

  T g1_y = -hy;  // g(y_q, y_ij)(-1)^I(x_q < x_ij)
  T g2_y = hy;
  T g3_y = -ly;
  T g4_y = ly;

  w_x = f1 * g1_y + f2 * g2_y + f3 * g3_y + f4 * g4_y;
  w_y = f1 * g1_x + f2 * g2_x + f3 * g3_x + f4 * g4_x;

  // reference in forward
  // T v1 = input[y_low * width + x_low];
  // T v2 = input[y_low * width + x_high];
  // T v3 = input[y_high * width + x_low];
  // T v4 = input[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename T>
__global__ void RoIAlignBackward(
    const int nthreads,
    const T* grad_output,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* grad_input,
    T* grad_bbox,
    const T* input,
    const T* rois,
    const int n_stride,
    const int c_stride,
    const int h_stride,
    const int w_stride) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_rois[1] * spatial_scale;
    T roi_start_h = offset_rois[2] * spatial_scale;
    T roi_end_w = offset_rois[3] * spatial_scale;
    T roi_end_h = offset_rois[4] * spatial_scale;

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_grad_input =
        grad_input + ((roi_batch_ind * channels + c) * height * width);

    // point to the current img feature map, the offset should be same as pointer "offset_grad_input"?
    const T* offset_input = input + ((roi_batch_ind * channels + c) * height * width);


    // We need to index the gradient using the tensor strides to access the
    // correct values.
    int output_offset = n * n_stride + c * c_stride;
    const T* offset_grad_output = grad_output + output_offset;
    const T grad_output_this_bin =
        offset_grad_output[ph * h_stride + pw * w_stride];

    // point to the current box's gradients
    // no index dim 0 for box grad
    T* offset_grad_bbox = grad_bbox + n * 4;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5

      T d_y = (y - roi_start_h) / roi_height;

      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T d_x = (x - roi_start_w) / roi_width;

        T w1, w2, w3, w4, w_x, w_y;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            offset_input,
            height,
            width,
            y,
            x,
            w1,
            w2,
            w3,
            w4,
            w_x,
            w_y,
            x_low,
            x_high,
            y_low,
            y_high,
            index);

        T g1 = grad_output_this_bin * w1 / count;
        T g2 = grad_output_this_bin * w2 / count;
        T g3 = grad_output_this_bin * w3 / count;
        T g4 = grad_output_this_bin * w4 / count;

        T g_x1 = grad_output_this_bin / count * w_x * (1 - d_x);
        T g_x2 = grad_output_this_bin / count * w_x * (d_x);
        T g_y1 = grad_output_this_bin / count * w_y * (1 - d_y);
        T g_y2 = grad_output_this_bin / count * w_y * (d_y);

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(
              offset_grad_input + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(
              offset_grad_input + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(
              offset_grad_input + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(
              offset_grad_input + y_high * width + x_high, static_cast<T>(g4));

          atomicAdd(offset_grad_bbox, static_cast<T>(g_x1));
          atomicAdd(offset_grad_bbox + 1, static_cast<T>(g_y1));
          atomicAdd(offset_grad_bbox + 2, static_cast<T>(g_x2));
          atomicAdd(offset_grad_bbox + 3, static_cast<T>(g_y2));

        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward

at::Tensor ROIAlign_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");

  at::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "ROIAlign_forward_cuda";
  at::checkAllSameGPU(c, {input_t, rois_t});
  at::checkAllSameType(c, {input_t, rois_t});

  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  at::Tensor output = at::zeros(
      {num_rois, channels, pooled_height, pooled_width}, input.options());

  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(output_size), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "ROIAlign_forward", [&] {
    RoIAlignForward<scalar_t><<<grid, block, 0, stream>>>(
        output_size,
        input.contiguous().data<scalar_t>(),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        rois.contiguous().data<scalar_t>(),
        output.data<scalar_t>());
  });
  AT_CUDA_CHECK(hipGetLastError());
  return output;
}

at::Tensor ROIAlign_backward_cuda(
    const at::Tensor& grad,
    const at::Tensor& rois,
    const at::Tensor& input,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int batch_size,
    const int channels,
    const int height,
    const int width,
    const int sampling_ratio,
    at::Tensor & grad_bbox2) {

  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");

  at::TensorArg grad_t{grad, "grad", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "ROIAlign_backward_cuda";
  at::checkAllSameGPU(c, {grad_t, rois_t});
  at::checkAllSameType(c, {grad_t, rois_t});

  at::cuda::CUDAGuard device_guard(grad.device());

  at::Tensor grad_input =
      at::zeros({batch_size, channels, height, width}, grad.options());

  int num_rois = rois.size(0);
  at::Tensor grad_bbox = at::zeros({num_rois, 4}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(grad.numel()), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  int n_stride = grad.stride(0);
  int c_stride = grad.stride(1);
  int h_stride = grad.stride(2);
  int w_stride = grad.stride(3);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.type(), "ROIAlign_backward", [&] {
    RoIAlignBackward<scalar_t><<<grid, block, 0, stream>>>(
        grad.numel(),
        grad.data<scalar_t>(),
        spatial_scale,
        channels,
        height,
        width,
        pooled_height,
        pooled_width,
        sampling_ratio,
        grad_input.data<scalar_t>(),
        grad_bbox.data<scalar_t>(),  // added
        input.contiguous().data<scalar_t>(),  // added
        rois.contiguous().data<scalar_t>(),
        n_stride,
        c_stride,
        h_stride,
        w_stride);
  });

  for(int i = 0; i < num_rois; i++ ){
    for(int j = 0; j < 4; j++)
    grad_bbox2[i][j] = grad_bbox[i][j];
  }

  AT_CUDA_CHECK(hipGetLastError());
  return grad_input;
}
